#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define DATA_SIZE 1048576
#define THREAD_NUM 256

int data[DATA_SIZE];

void GenerateNumbers(int *number, int size)
{
    for (int i = 0; i < size; i++) {
        number[i] = rand() % 10;
    }
}

void printDeviceProp(const hipDeviceProp_t &prop)
{
    printf("Device Name : %s.\n", prop.name);
    printf("totalGlobalMem : %ld.\n", prop.totalGlobalMem);
    printf("sharedMemPerBlock : %ld.\n", prop.sharedMemPerBlock);
    printf("regsPerBlock : %d.\n", prop.regsPerBlock);
    printf("warpSize : %d.\n", prop.warpSize);
    printf("memPitch : %ld.\n", prop.memPitch);
    printf("maxThreadsPerBlock : %d.\n", prop.maxThreadsPerBlock);
    printf("maxThreadsDim[0 - 2] : %d %d %d.\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    printf("maxGridSize[0 - 2] : %d %d %d.\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    printf("totalConstMem : %ld.\n", prop.totalConstMem);
    printf("major.minor : %d.%d.\n", prop.major, prop.minor);
    printf("clockRate : %d.\n", prop.clockRate);
    printf("textureAlignment : %ld.\n", prop.textureAlignment);
    printf("deviceOverlap : %d.\n", prop.deviceOverlap);
    printf("multiProcessorCount : %d.\n", prop.multiProcessorCount);

}

bool InitCUDA()
{
    int count;

    hipGetDeviceCount(&count);

    if (count == 0) {
        fprintf(stderr, "There is no device.\n");
        return false;
    }

    int i;
    for (i = 0; i < count; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printDeviceProp(prop);
        if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
            if (prop.major >= 1)
                break;
        }
    }
    if (i == count) {
        fprintf(stderr, "There is no device supporting CUDA 1.X. \n");
        return false;
    }

    hipSetDevice(i);
    return true;
}

__global__ void sumOfSquares(int *num, int *result, clock_t *time)
{
    // 表示当前线程是第几个thread
    int tid = threadIdx.x;
    // 计算每个线程需要完成的量
    int size = DATA_SIZE / THREAD_NUM;
    int sum = 0;
    int i;
    clock_t start ;
    if (tid == 0) {
        start = clock();
    }
    for (i = tid; i < DATA_SIZE; i += THREAD_NUM) {
        sum += num[i] * num[i] * num[i];
    }
    /*
    for (i = 0; i < DATA_SIZE; i++) {
        sum += num[i] * num[i] * num[i];
    }
    */
    *result = sum;
    if (tid == 0)
        *time = clock() - start;
}

int main()
{
    if (!InitCUDA()) {
        return 0;
    }
    printf("CUDA initialized.\n");

    GenerateNumbers(data, DATA_SIZE);

    int *gpudata, *result;
    clock_t *time;

    hipMalloc((void **)&gpudata, sizeof(int)*DATA_SIZE);
    hipMalloc((void **)&result, sizeof(int)*THREAD_NUM);
    hipMalloc((void **)&time, sizeof(clock_t));

    hipMemcpy(gpudata, data, sizeof(int)*DATA_SIZE, hipMemcpyHostToDevice);

    sumOfSquares<<<1, THREAD_NUM, 0>>>(gpudata, result, time);

    int sum[THREAD_NUM];
    clock_t time_used;
    hipMemcpy(&sum, result, sizeof(int)*THREAD_NUM, hipMemcpyDeviceToHost);
    hipMemcpy(&time_used, time, sizeof(clock_t), hipMemcpyDeviceToHost);

    hipFree(gpudata);
    hipFree(result);
    hipFree(time);

    int final_sum = 0;
    for (int i = 0; i < THREAD_NUM; i++) {
        final_sum += sum[i];
    }

    printf("GPU sum: %d, time used: %ld\n", final_sum, time_used);

    final_sum = 0;
    time_used = clock();
    for (int i = 0; i < DATA_SIZE; i++) {
        final_sum += data[i] * data[i] * data[i];
    }
    time_used = clock() - time_used;

    printf("CPU sum: %d, time used: %ld\n", final_sum, time_used);
    return 0;
}
